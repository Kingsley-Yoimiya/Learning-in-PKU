#include <stdio.h>
#include <stdlib.h>
#include "reduction.cuh"

/*
 * Main program for benchmarking different CUDA reduction implementations
 * Compares performance of reduction kernels with baseline GPU kernel
 */

int main(int argc, char **argv) {
    // Allocate and initialize host memory
    int *h_input = (int *)malloc(MEM_SIZE);
    int gold, h_output = 0;  // gold stores CPU reference result
    initData(h_input, VEC_LEN, &gold);  // Initialize input data

    // Allocate device memory
    int *d_input, *d_output;
    checkCuda(hipMalloc(&d_input, MEM_SIZE));  // Allocate input array on device
    checkCuda(hipMalloc(&d_output, sizeof(int)));  // Allocate output scalar on device
    checkCuda(hipMemcpy(d_input, h_input, MEM_SIZE, hipMemcpyHostToDevice));  // Copy data to device

    // Create CUDA events for timing measurements
    hipEvent_t startEvent, stopEvent;
    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));
    float duration;  // Will store elapsed time in milliseconds

    // Print benchmark header
    printf("================================================================\n");
    printf("%30s%30s\n", "Routine", "Reduction time (ms)");

    /* Benchmark naive reduction kernel */
    printf("%30s", "Baseline reduction");
    // Configure kernel launch parameters:
    // - Each block has 1 thread
    // - Each thread processes NAIVE_BLOCK_DIM elements
    // - Total blocks = VEC_LEN / NAIVE_BLOCK_DIM
    dim3 dimBlockNaive(1);
    dim3 dimGridNaive(VEC_LEN / NAIVE_BLOCK_DIM);
    
    // Warmup run to avoid startup timing artifacts
    checkCuda(hipMemset(d_output, 0, sizeof(int)));  // Clear output
    reductionNaive<<<dimGridNaive, dimBlockNaive>>>(d_output, d_input, VEC_LEN);

    // Time the kernel execution
    checkCuda(hipEventRecord(startEvent, 0));  // Start timer
    for (int i = 0; i < NUM_REPS; i++) {
        checkCuda(hipMemset(d_output, 0, sizeof(int)));  // Reset output
        reductionNaive<<<dimGridNaive, dimBlockNaive>>>(d_output, d_input, VEC_LEN);
    }
    checkCuda(hipEventRecord(stopEvent, 0));  // Stop timer
    checkCuda(hipEventSynchronize(stopEvent));  // Wait for stop event
    checkCuda(hipEventElapsedTime(&duration, startEvent, stopEvent));  // Get duration
    
    // Copy result back to host and verify
    checkCuda(hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost));
    postProcess(&gold, &h_output, duration / NUM_REPS);  // Compare with CPU result

    // Your optimized kernel calls go here

    printf("================================================================\n");

    // Clean up allocated resources
    free(h_input);
    checkCuda(hipFree(d_input));
    checkCuda(hipFree(d_output));
    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));
}
