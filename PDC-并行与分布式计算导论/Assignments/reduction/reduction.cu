#include "hip/hip_runtime.h"
#include "reduction.cuh"

// naive reduction kernel
// This kernel is barely optimized and serves as a baseline
__global__ void reductionNaive(int *output, const int *input,
                               const int length) {
    int i = blockIdx.x * NAIVE_BLOCK_DIM;
    int partialSum = 0;
    
    for (int j = 0; j < NAIVE_BLOCK_DIM; j++) {
        if (i + j < length) {
            partialSum += input[i + j];
        }
    }
    atomicAdd(output, partialSum);
}

// Your optimized kernel implementations go here
// __global__ void reductionOptimized(int *output, const int *input, const int length) {}